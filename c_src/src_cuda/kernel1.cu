
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include <CUDA.h>
//#include <math.h>



#define SIZE	512

__global__ void iterate(float* U, float* Unew)
{
	int m = blockIdx.x*SIZE+threadIdx.x+SIZE+1;
	Unew[m]=0.25*(U[m-SIZE]+U[m+SIZE]+U[m-1]+U[m+1]);
}


int main()
{
	float *U;
	float *dU, *dUnew;
	double h=M_PI/SIZE;
	int n=0,i=0;

	U = (float *)malloc(SIZE*SIZE*sizeof(float));

	hipMalloc( &dU, 	SIZE*SIZE*sizeof(float));
	hipMalloc( &dUnew, 	SIZE*SIZE*sizeof(float));
	
	
	
	for(i=0;i<SIZE;i++)
	{
		U[i]=sin(i*h);
		U[(SIZE-1)*SIZE+i]=U[i];
		U[i*SIZE]=-U[i];
		U[i*SIZE+SIZE-1]=-U[i];
	}


	hipMemcpy( dU, 	U, SIZE*SIZE*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dUnew, 	U, SIZE*SIZE*sizeof(float), hipMemcpyHostToDevice );



	for(n=0;n<100000;n++){
		iterate<<<SIZE-2,SIZE-2>>>(dU, dUnew);
		hipMemcpy( dU, dUnew, SIZE*SIZE*sizeof(float), hipMemcpyDeviceToDevice) ;
	}
	
	
	
	hipMemcpy( U, dU, SIZE*SIZE*sizeof(float), hipMemcpyDeviceToHost );


	for(i=0;i<SIZE*SIZE;i+=30)
	{
		printf("%f %f %f\n",h*(i%SIZE),h*(i/SIZE),U[i]);
	}
	
	
	
	free(U);
	hipFree(dU);
	hipFree(dUnew);

	return 0;
}
