#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gsl/gsl_linalg.h>


#define N 10
#define X0 -1.
#define X1  1.
#define Y0 -1.
#define Y1  1.

//depends on which kind of variable supports patricular GPU

//16-bit (tested on CUDA 5.5 + GT8600M + OS X 10.9)
#define diff_step 0.0078125
#define glob_delta 128.

//32-bit (tested on CUDA 5.5 + GTX660 + xUbuntu)
//#define diff_step 0.0009765625
//#define glob_delta 1024.

//64-bit (untested on GPU, tested only on CPU)
//#define diff_step 0.00006103515625
//#define glob_delta 16384.
#define intStep 1.

typedef struct basis_args {
    double	x, y;
    int		m, n;
} basis_args;

typedef struct rect_area {
    double x0, x1;
    double y0, y1;
} rect_area;

__device__ double phi (double x, double y, int n)
{
    return pow(x,n%N)*pow(y,n/N);
}
__device__ double omega(double x, double y) {
    return (x-X0)*(x-X1)*(y-Y0)*(y-Y1);
}
__device__ double structure(double x, double y, int n)
{
    return phi(x,y,n)*omega(x,y);
}

__host__ double Hphi (double x, double y, int n)
{
    return pow(x,n%N)*pow(y,n/N);
}
__host__ double Homega(double x, double y) {
    return (x-X0)*(x-X1)*(y-Y0)*(y-Y1);
}
__host__ double Hstructure(double x, double y, int n)
{
    return Hphi(x,y,n)*Homega(x,y);
}
__host__ double reconstruct_at(gsl_vector *solution,
                               double x, double y)
// Reconstucts value of solution at point (x,y)
{
    int i;
    double result = 0.;
    for(i=0; i<N*N; i++)
        result += gsl_vector_get(solution, i)*Hstructure(x,y,i);

    return result;
}

void plot_region(gsl_vector *solution/*, rect_area plot_area*/)
{
    double hx = (X1-X0)/64.,
           hy = (Y1-Y0)/64.,
           i,j;

    FILE * op;
    op = fopen("../plot_data/plot_region.txt", "w");
    for(i=X0; i<=X1; i+=hx)
        for(j=Y0; j<=Y1; j+=hy)
        {
            fprintf(op, "%15.15f,%15.15f,%15.15f;\n", i,j, reconstruct_at(solution,i,j));
        }
    fclose(op);
    i = system("../bin/plotter.py ../plot_data/plot_region Numerical &");
}


__device__ double right_part_f(double x, double y)
{
    return 12.*(y*y*(x*x*x*x-1.) + x*x*(y*y*y*y-1.));
}


__device__ double left_under_int_new(basis_args arguments)
{
    double 	x = arguments.x;
    double 	y = arguments.y;
    int 	m = arguments.m;
    int 	n = arguments.n;

    return  	structure(x,y,m)*(
                    structure(x+diff_step,y,n)+structure(x-diff_step,y,n)+
                    structure(x,y+diff_step,n)+structure(x,y-diff_step,n)
                    -4.*structure(x,y,n))*glob_delta*glob_delta;
}
__device__ double right_under_int_new(basis_args arguments)
{
    double 	x = arguments.x;
    double 	y = arguments.y;
    int 	m = arguments.m;

    return right_part_f(x,y)*structure(x,y,m);
}


__device__ double nodes[16], weights[16];
__global__ void initGaussInt()
{
    //initializing node & weights
    nodes[0] 	= -0.0950125098376374;
    nodes[1] 	=  0.0950125098376374;
    nodes[2] 	= -0.2816035507792589;
    nodes[3] 	=  0.2816035507792589;
    nodes[4] 	= -0.4580167776572274;
    nodes[5] 	=  0.4580167776572274;
    nodes[6] 	= -0.6178762444026438;
    nodes[7] 	=  0.6178762444026438;
    nodes[8] 	= -0.7554044083550030;
    nodes[9] 	=  0.7554044083550030;
    nodes[10] 	= -0.8656312023878318;
    nodes[11] 	=  0.8656312023878318;
    nodes[12] 	= -0.9445750230732326;
    nodes[13]	=  0.9445750230732326;
    nodes[14] 	= -0.9894009349916499;
    nodes[15] 	=  0.9894009349916499;


    weights[0] 	=  0.1894506104550685;
    weights[1] 	=  0.1894506104550685;
    weights[2] 	=  0.1826034150449236;
    weights[3] 	=  0.1826034150449236;
    weights[4] 	=  0.1691565193950025;
    weights[5] 	=  0.1691565193950025;
    weights[6] 	=  0.1495959888165767;
    weights[7] 	=  0.1495959888165767;
    weights[8] 	=  0.1246289712555339;
    weights[9] 	=  0.1246289712555339;
    weights[10] =  0.0951585116824928;
    weights[11] =  0.0951585116824928;
    weights[12] =  0.0622535239386479;
    weights[13] =  0.0622535239386479;
    weights[14] =  0.0271524594117541;
    weights[15] =  0.0271524594117541;
}

__device__ double gauss_integral_right1(rect_area int_area,
                                       basis_args args
                                       )
{
    double x0 = int_area.x0;
    double x1 = int_area.x1;

    int i,j;
    double res = 0., stepx = (x1-x0)/intStep;

    basis_args temp_args = args;

        for (i = 1; i <= intStep; i++)
        {
            for (j = 0; j < 16; j++)
            {
                temp_args.x = (double)(i-1)*stepx + x0 + 0.5*(nodes[j]+1.)*stepx;
                res += weights[j]*right_under_int_new(temp_args);
            }
        }

        return 0.5*res*stepx;
}

__device__ double gauss_integral_right2(rect_area int_area,
                                       basis_args args
                                       )
{
    double y0 = int_area.y0;
    double y1 = int_area.y1;

    int i,j;
    double res = 0., stepy = (y1-y0)/intStep;

    basis_args temp_args = args;

        for (i = 1; i <= intStep; i++)
        {
            for (j = 0; j < 16; j++)
            {
                temp_args.y = (double)(i-1)*stepy + y0 + 0.5*(nodes[j]+1.)*stepy;
                res += weights[j]*gauss_integral_right1( int_area, temp_args);
            }
        }

        return 0.5*res*stepy;
}

__device__ double gauss_integral_left1(rect_area int_area,
                                       basis_args args
                                       )
{
    double x0 = int_area.x0;
    double x1 = int_area.x1;

    int i,j;
    double res = 0., stepx = (x1-x0)/intStep;

    basis_args temp_args = args;

    for (i = 1; i <= intStep; i++)
    {
        for (j = 0; j < 16; j++)
        {
            temp_args.x = (double)(i-1)*stepx + x0 + 0.5*(nodes[j]+1.)*stepx;
            res += weights[j]*left_under_int_new(temp_args);
        }
    }

    return 0.5*res*stepx;
}
__device__ double gauss_integral_left2(rect_area int_area,
                                       basis_args args
                                       )
{
    double y0 = int_area.y0;
    double y1 = int_area.y1;

    int i,j;
    double res = 0., stepy = (y1-y0)/intStep;

    basis_args temp_args = args;

    for (i = 1; i <= intStep; i++)
    {
        for (j = 0; j < 16; j++)
        {
            temp_args.y = (double)(i-1)*stepy + y0 + 0.5*(nodes[j]+1.)*stepy;
            res += weights[j]*gauss_integral_left1( int_area, temp_args);
        }
    }

    return 0.5*res*stepy;
}





__global__ void form_matrix_new (float *sys,
                                 float *RightPart)
{
    int i = blockIdx.x, j = threadIdx.x;
    basis_args args;
    args.x = 0.;
    args.y = 0.;
    args.m = 0;
    args.n = 0;
	rect_area int_area = {.x0 = X0, .x1 = X1, .y0 = Y0, .y1 = Y1};
	
    args.m = i;
    
    RightPart[i] = gauss_integral_right2(int_area,args);
    
    args.n = j;
    
    
    sys[i*N*N+j] = gauss_integral_left2(int_area, args);
}


int main()
{
    initGaussInt<<<1,1>>>();

    //as usual, we define pointer to arrays in RAM and GPU RAM
    float *System, *right_part;//, *solution;
    float *dev_System, *dev_right_part;//, *dev_solution;

    //and allocaing this memory
    System = (float *)malloc(N*N * N*N*sizeof(float));
    hipMalloc (&dev_System, N*N * N*N*sizeof(float));
    right_part = (float *)malloc(N*N*sizeof(float));
    hipMalloc (&dev_right_part, N*N*sizeof(float));

    //forming the system. TODO: rename this func...
    form_matrix_new<<<N*N, N*N>>>(dev_System, dev_right_part);
    hipMemcpy( System, dev_System, N*N * N*N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( right_part, dev_right_part, N*N*sizeof(float), hipMemcpyDeviceToHost);


    gsl_matrix 	*Gsys;
    gsl_vector  *Grightpart, *Gsolution;
    Gsys		= gsl_matrix_alloc (N*N,N*N);
    Grightpart	= gsl_vector_alloc(N*N);
    Gsolution	= gsl_vector_alloc(N*N);
    int i, j;

    //filling library-specified mathematical objects
    for(i = 0; i < N*N; i++)
    {
        gsl_vector_set(Grightpart, i, right_part[i]);
        for(j = 0; j < N*N; j++)
        {
            gsl_matrix_set(Gsys, i,j, System[i*N*N+j]);
        }
    }

    gsl_permutation * p = gsl_permutation_alloc (N*N);
    gsl_linalg_LU_decomp (Gsys, p, &i);
    gsl_linalg_LU_solve (Gsys, p, Grightpart, Gsolution);

	plot_region(Gsolution);


    return 0;
}
