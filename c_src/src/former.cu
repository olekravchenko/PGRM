#include "hip/hip_runtime.h"
#include <stdio.h>
#include <gsl/gsl_linalg.h>


#define N 10
#define X0 -1.
#define X1  1.
#define Y0 -1.
#define Y1  1.

//depends on which kind of variable supports patricular GPU

//16-bit (tested on CUDA 5.5 + GT8600M + OS X 10.9)
//#define diff_step 0.0078125
//#define glob_delta 128.

//32-bit (tested on CUDA 5.5 + GTX660 + xUbuntu)
//#define diff_step 0.0009765625
//#define glob_delta 1024.

//64-bit (untested on GPU, tested only on CPU)
#define diff_step 0.00006103515625
#define glob_delta 16384.
#define intStep 1.

typedef struct basis_args {
    double	x, y;
    int		m, n;
} basis_args;

typedef struct rect_area {
    double x0, x1;
    double y0, y1;
} rect_area;


#include "task.cu"



__host__ double reconstruct_at(gsl_vector *solution,
                               double x, double y)
{
    int i;
    double result = 0.;
    for(i=0; i<N*N; i++)
        result += gsl_vector_get(solution, i)*Hstructure(x,y,i);

    return result;
}

void plot_region(gsl_vector *solution/*, rect_area plot_area*/)
{
    double hx = (X1-X0)/64.,
           hy = (Y1-Y0)/64.,
           i,j;

    FILE * op;
    op = fopen("../plot_data/plot_region", "w");
    for(i=X0; i<=X1; i+=hx)
        for(j=Y0; j<=Y1; j+=hy)
            fprintf(op, "%15.15f %15.15f %15.15f\n", i,j, reconstruct_at(solution,i,j));
    fclose(op);
    i = system("screen -d -m ../bin/Plot &");
}


__device__ double left_under_int_new(basis_args arguments)
{
    double 	x = arguments.x;
    double 	y = arguments.y;
    int 	m = arguments.m;
    int 	n = arguments.n;

	//returns S\phi_m \Delta (S\phi_n), where S is operator of the structure
    return			structure(x,y,m)*(
                    structure(x+diff_step,y,n)+structure(x-diff_step,y,n)+
                    structure(x,y+diff_step,n)+structure(x,y-diff_step,n)
                    -4.*structure(x,y,n))*glob_delta*glob_delta;
}
__device__ double right_under_int_new(basis_args arguments)
{
    double 	x = arguments.x;
    double 	y = arguments.y;
    int 	m = arguments.m;

    return right_part_f(x,y)*structure(x,y,m);
}


__device__ double nodes[16], weights[16];
__global__ void initGaussInt()
//initialization of node & weights values
{
    nodes[0] 	= -0.0950125098376374;
    nodes[1] 	=  0.0950125098376374;
    nodes[2] 	= -0.2816035507792589;
    nodes[3] 	=  0.2816035507792589;
    nodes[4] 	= -0.4580167776572274;
    nodes[5] 	=  0.4580167776572274;
    nodes[6] 	= -0.6178762444026438;
    nodes[7] 	=  0.6178762444026438;
    nodes[8] 	= -0.7554044083550030;
    nodes[9] 	=  0.7554044083550030;
    nodes[10] 	= -0.8656312023878318;
    nodes[11] 	=  0.8656312023878318;
    nodes[12] 	= -0.9445750230732326;
    nodes[13]	=  0.9445750230732326;
    nodes[14] 	= -0.9894009349916499;
    nodes[15] 	=  0.9894009349916499;


    weights[0] 	=  0.1894506104550685;
    weights[1] 	=  0.1894506104550685;
    weights[2] 	=  0.1826034150449236;
    weights[3] 	=  0.1826034150449236;
    weights[4] 	=  0.1691565193950025;
    weights[5] 	=  0.1691565193950025;
    weights[6] 	=  0.1495959888165767;
    weights[7] 	=  0.1495959888165767;
    weights[8] 	=  0.1246289712555339;
    weights[9] 	=  0.1246289712555339;
    weights[10] =  0.0951585116824928;
    weights[11] =  0.0951585116824928;
    weights[12] =  0.0622535239386479;
    weights[13] =  0.0622535239386479;
    weights[14] =  0.0271524594117541;
    weights[15] =  0.0271524594117541;
}




/*
 * TODO: try to reduce quantity of similar gauss integral functions, as next to the CPU
 *  way as only possible
 */


__device__ double gauss_integral_right1(rect_area int_area,
                                       basis_args args)
{
    double x0 = int_area.x0;
    double x1 = int_area.x1;

    int i,j;
    double res = 0., stepx = (x1-x0)/intStep;

    basis_args temp_args = args;

        for (i = 1; i <= intStep; i++)
        {
            for (j = 0; j < 16; j++)
            {
                temp_args.x = (double)(i-1)*stepx + x0 + 0.5*(nodes[j]+1.)*stepx;
                res += weights[j]*right_under_int_new(temp_args);
            }
        }

        return 0.5*res*stepx;
}

__device__ double gauss_integral_right2(rect_area int_area,
                                       basis_args args)
{
    double y0 = int_area.y0;
    double y1 = int_area.y1;

    int i,j;
    double res = 0., stepy = (y1-y0)/intStep;

    basis_args temp_args = args;

        for (i = 1; i <= intStep; i++)
        {
            for (j = 0; j < 16; j++)
            {
                temp_args.y = (double)(i-1)*stepy + y0 + 0.5*(nodes[j]+1.)*stepy;
                res += weights[j]*gauss_integral_right1( int_area, temp_args);
            }
        }

        return 0.5*res*stepy;
}

__device__ double gauss_integral_left1(rect_area int_area,
                                       basis_args args)
{
    double x0 = int_area.x0;
    double x1 = int_area.x1;

    int i,j;
    double res = 0., stepx = (x1-x0)/intStep;

    basis_args temp_args = args;

    for (i = 1; i <= intStep; i++)
    {
        for (j = 0; j < 16; j++)
        {
            temp_args.x = (double)(i-1)*stepx + x0 + 0.5*(nodes[j]+1.)*stepx;
            res += weights[j]*left_under_int_new(temp_args);
        }
    }

    return 0.5*res*stepx;
}
__device__ double gauss_integral_left2(rect_area int_area,
                                       basis_args args)
{
    double y0 = int_area.y0;
    double y1 = int_area.y1;

    int i,j;
    double res = 0., stepy = (y1-y0)/intStep;

    basis_args temp_args = args;

    for (i = 1; i <= intStep; i++)
    {
        for (j = 0; j < 16; j++)
        {
            temp_args.y = (double)(i-1)*stepy + y0 + 0.5*(nodes[j]+1.)*stepy;
            res += weights[j]*gauss_integral_left1( int_area, temp_args);
        }
    }

    return 0.5*res*stepy;
}





__global__ void form_PGRM_sle (float *sys, float *RightPart)
{
    int i = blockIdx.x, j = threadIdx.x;
    basis_args args;
    args.x = 0.;
    args.y = 0.;
    args.m = 0;
    args.n = 0;
	rect_area int_area = {.x0 = X0, .x1 = X1, .y0 = Y0, .y1 = Y1};
	
    args.m = i;
    
    RightPart[i] = gauss_integral_right2(int_area,args);
    
    args.n = j;
    
    
    sys[i*N*N+j] = gauss_integral_left2(int_area, args);
}

__host__ void solve_sle_with_reconstruction(float *System, float *right_part)
{
    gsl_matrix 	*Gsys = gsl_matrix_alloc (N*N,N*N);
    gsl_vector  *Grightpart = gsl_vector_alloc(N*N), 
				*Gsolution = gsl_vector_alloc(N*N);
    int i, j;

	//filling library-specified mathematical objects
    for(i = 0; i < N*N; i++)
    {
        gsl_vector_set(Grightpart, i, right_part[i]);
        for(j = 0; j < N*N; j++)
        {
            gsl_matrix_set(Gsys, i,j, System[i*N*N+j]);
        }
    }

    gsl_permutation * p = gsl_permutation_alloc (N*N);
    gsl_linalg_LU_decomp (Gsys, p, &i);
    gsl_linalg_LU_solve (Gsys, p, Grightpart, Gsolution);

	plot_region(Gsolution);
}

__host__ void form_sle_on_gpu (float *System, float *right_part)
{
	float *dev_System, *dev_right_part;

	hipMalloc (&dev_System, N*N * N*N*sizeof(float));
    hipMalloc (&dev_right_part, N*N*sizeof(float));

    form_PGRM_sle<<<N*N, N*N>>>(dev_System, dev_right_part);

	hipMemcpy( System, dev_System, N*N * N*N*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy( right_part, dev_right_part, N*N*sizeof(float), hipMemcpyDeviceToHost);
	
	hipFree(dev_right_part);
	hipFree(dev_System);
}

int main()
{
    initGaussInt<<<1,1>>>();
	//phi = &phi_b3;
    set_b3<<<1,1>>>();
	
	//as usual, we define pointer to arrays in RAM and GPU RAM
    float *System, *right_part;
    System = (float *)malloc(N*N * N*N*sizeof(float));
    right_part = (float *)malloc(N*N*sizeof(float));

	form_sle_on_gpu (System, right_part);
	solve_sle_with_reconstruction(System, right_part);
	
    return 0;
}
