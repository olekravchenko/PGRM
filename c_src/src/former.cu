
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 	10
#define X0 -1.
#define X1  1.
#define Y0 -1.
#define Y1  1.
#define diff_step 0.00000001
#define glob_delta 100000000.
#define intStep 2.

typedef struct basis_args {
    double	x, y;
    int		m, n;
} basis_args;

typedef struct rect_area {
    double x0, x1;
    double y0, y1;
} rect_area;

__device__ double phi (double x, double y, int n)
{
    return pow(x,n%N)*pow(y,n/N);
}
__device__ double omega(double x, double y) {
    return (x-X0)*(x-X1)*(y-Y0)*(y-Y1);
}
__device__ double structure(double x, double y, int n)
{
    return phi(x,y,n)*omega(x,y);
}
__device__ double right_part_f(double x, double y)
{
    return 12.*(y*y*(x*x*x*x-1.) + x*x*(y*y*y*y-1.));
}


__device__ double left_under_int_new(basis_args arguments)
{
    double 	x = arguments.x;
    double 	y = arguments.y;
    int 	m = arguments.m;
    int 	n = arguments.n;

    return  	structure(x,y,m)*(
                    structure(x+diff_step,y,n)+structure(x-diff_step,y,n)+
                    structure(x,y+diff_step,n)+structure(x,y-diff_step,n)
                    -4.*structure(x,y,n))*glob_delta*glob_delta;
}
__device__ double right_under_int_new(basis_args arguments)
{
    double 	x = arguments.x;
    double 	y = arguments.y;
    int 	m = arguments.m;

    return right_part_f(x,y)*structure(x,y,m);
}


__device__ double nodes[16], weights[16];
__global__ void initGaussInt()
{
    //initializing node & weights
    nodes[0] 	= -0.0950125098376374;
    nodes[1] 	=  0.0950125098376374;
    nodes[2] 	= -0.2816035507792589;
    nodes[3] 	=  0.2816035507792589;
    nodes[4] 	= -0.4580167776572274;
    nodes[5] 	=  0.4580167776572274;
    nodes[6] 	= -0.6178762444026438;
    nodes[7] 	=  0.6178762444026438;
    nodes[8] 	= -0.7554044083550030;
    nodes[9] 	=  0.7554044083550030;
    nodes[10] 	= -0.8656312023878318;
    nodes[11] 	=  0.8656312023878318;
    nodes[12] 	= -0.9445750230732326;
    nodes[13]	=  0.9445750230732326;
    nodes[14] 	= -0.9894009349916499;
    nodes[15] 	=  0.9894009349916499;


    weights[0] 	=  0.1894506104550685;
    weights[1] 	=  0.1894506104550685;
    weights[2] 	=  0.1826034150449236;
    weights[3] 	=  0.1826034150449236;
    weights[4] 	=  0.1691565193950025;
    weights[5] 	=  0.1691565193950025;
    weights[6] 	=  0.1495959888165767;
    weights[7] 	=  0.1495959888165767;
    weights[8] 	=  0.1246289712555339;
    weights[9] 	=  0.1246289712555339;
    weights[10] =  0.0951585116824928;
    weights[11] =  0.0951585116824928;
    weights[12] =  0.0622535239386479;
    weights[13] =  0.0622535239386479;
    weights[14] =  0.0271524594117541;
    weights[15] =  0.0271524594117541;
}

__device__ double gauss_integral_right1(rect_area int_area,
                                       basis_args args
                                       )
{
    double x0 = int_area.x0;
    double x1 = int_area.x1;
    //~ double y0 = int_area.y0;
    //~ double y1 = int_area.y1;

    int i,j;
    double res = 0., stepx = (x1-x0)/intStep;//, stepy = (y1-y0)/intStep;

    basis_args temp_args = args;

    //integral calculations
        for (i = 1; i <= intStep; i++)
        {
            for (j = 0; j < 16; j++)
            {
                temp_args.x = (double)(i-1)*stepx + x0 + 0.5*(nodes[j]+1.)*stepx;
                res += weights[j]*right_under_int_new(temp_args);
            }
        }

        return 0.5*res*stepx;
}

__device__ double gauss_integral_right2(rect_area int_area,
                                       basis_args args
                                       )
{
    //~ double x0 = int_area.x0;
    //~ double x1 = int_area.x1;
    double y0 = int_area.y0;
    double y1 = int_area.y1;

    int i,j;
    double res = 0.,/* stepx = (x1-x0)/intStep, */stepy = (y1-y0)/intStep;

    basis_args temp_args = args;

    //integral calculations
        for (i = 1; i <= intStep; i++)
        {
            for (j = 0; j < 16; j++)
            {
                temp_args.y = (double)(i-1)*stepy + y0 + 0.5*(nodes[j]+1.)*stepy;
                //res += weights[j]*SubIntegralLeft((*f),x0,x1,(double)(i-1)*step + x0 + 0.5*(nodes[j]+1.)*step,k1,k2);
                res += weights[j]*gauss_integral_right1( int_area, temp_args);
            }
        }

        return 0.5*res*stepy;
}

__device__ double gauss_integral_left1(rect_area int_area,
                                       basis_args args
                                       )
{
    double x0 = int_area.x0;
    double x1 = int_area.x1;
    //~ double y0 = int_area.y0;
    //~ double y1 = int_area.y1;

    int i,j;
    double res = 0., stepx = (x1-x0)/intStep;//, stepy = (y1-y0)/intStep;

    basis_args temp_args = args;

    //integral calculations
    for (i = 1; i <= intStep; i++)
    {
        for (j = 0; j < 16; j++)
        {
            temp_args.x = (double)(i-1)*stepx + x0 + 0.5*(nodes[j]+1.)*stepx;
            res += weights[j]*left_under_int_new(temp_args);
        }
    }

    return 0.5*res*stepx;
}
__device__ double gauss_integral_left2(rect_area int_area,
                                       basis_args args
                                       )
{
    //~ double x0 = int_area.x0;
    //~ double x1 = int_area.x1;
    double y0 = int_area.y0;
    double y1 = int_area.y1;

    int i,j;
    double res = 0.,/* stepx = (x1-x0)/intStep, */stepy = (y1-y0)/intStep;

    basis_args temp_args = args;

    //integral calculations
    for (i = 1; i <= intStep; i++)
    {
        for (j = 0; j < 16; j++)
        {
            temp_args.y = (double)(i-1)*stepy + y0 + 0.5*(nodes[j]+1.)*stepy;
            //res += weights[j]*SubIntegralLeft((*f),x0,x1,(double)(i-1)*step + x0 + 0.5*(nodes[j]+1.)*step,k1,k2);
            res += weights[j]*gauss_integral_left1( int_area, temp_args);
        }
    }

    return 0.5*res*stepy;
}





__global__ void form_matrix_new (float *sys)/*,
                                 //float *RightPart,
                                 rect_area int_area)*/
{
    int i = blockIdx.x, j = threadIdx.x;
    basis_args args;
    args.x = 0.;
    args.y = 0.;
    args.m = 0;
    args.n = 0;
	rect_area int_area = {.x0 = X0, .x1 = X1, .y0 = Y0, .y1 = Y1};
    //~ for(i = 0; i < N*N; i++) // replace this two loops with links to blockId and threadId
    //~ {
    args.m = i;
    
    //RightPart[i] = gauss_integral_right2(int_area,args);
    
    
    //~ for(j = 0; j < N*N; j++)
    //~ {
    args.n = j;
    sys[i*N*N+j] = gauss_integral_left2(int_area, args);
    //~ }
    //~ }
}
//used as example from previous project
__global__ void iter		(float *U, float *Unew, int size)
{
    int k = (blockIdx.x + 1)*size + (threadIdx.x +1);
    float h=0.01;
    Unew[k] = 0.25*(U[k+size]+U[k-size]+U[k-1]+U[k+1]-h*h*2*expf(h*(blockIdx.x+threadIdx.x+2)));
}


int main()
{
    //pointers to host arrays
    float *System;//, *right_part, *solution;
	//rect_area *Area;
    //pointers to device copies of host arrays
    float *dev_System;//, *dev_right_part, *dev_solution;
	
    System = (float *)malloc(N*N * N*N*sizeof(float));
    hipMalloc( &dev_System, N*N * N*N*sizeof(float));
	int i;//,j;
	for(i = 0; i< N*N * N*N; i++)
	{
		System[i] = 0.;
	}
	hipMemcpy( dev_System, System, N*N * N*N*sizeof(float), hipMemcpyHostToDevice);
	form_matrix_new<<<N*N, N*N>>>(dev_System);
	hipMemcpy( System, dev_System, N*N * N*N*sizeof(float), hipMemcpyDeviceToHost);
	for(i = 0; i < N*N * N*N; i++)
	{
		printf("%3.3f ",System[i]);
		if(i%(N*N) == 0)
			printf("\n");
	}
	printf("\n");
    return 0;
}
