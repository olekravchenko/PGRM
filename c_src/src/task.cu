#include "hip/hip_runtime.h"

__device__ double dev_f_B_3(double X)
{
	double absV=fabs(X);
	if(absV<2)
	{
		if(absV>=1)
			return 0.25*(2.0-absV)*(2.0-absV)*(2.0-absV); 
		else
			return 1.0 - 1.5*absV*absV*(1.0 - 0.5*absV);
	}
	return 0.0;
}

__host__ double host_f_B_3(double X)
{
	double absV=fabs(X);
	if(absV<2)
	{
		if(absV>=1)
		{
			return 0.25*(2.0-absV)*(2.0-absV)*(2.0-absV); 
		}
		else
			return 1.0 - 1.5*absV*absV*(1.0 - 0.5*absV);
	}
	return 0.0;
}


__device__ double (*phi)(double, double, int);

__device__ double phi_b3 (double x, double y, int n)
{
	//polynomial basis
//     return pow(x,n%N)*pow(y,n/N);
	
	double cubic_stepx = (X1-X0)/(double)(N-1);
	double cubic_stepy = (Y1-Y0)/(double)(N-1);
	
	return dev_f_B_3((N-1)/(X1-X0)*(x-X0-cubic_stepx*(double)(n%(N))))*
           dev_f_B_3((N-1)/(Y1-Y0)*(y-Y0-cubic_stepy*(double)(n/(N))));
}

__global__ void set_b3()
{
	phi = &phi_b3;
	
}

__device__ double omega(double x, double y) 
{
    return (x-X0)*(x-X1)*(y-Y0)*(y-Y1);
}
__device__ double structure(double x, double y, int n)
{
	//structure for 1st boundary problem
    return phi(x,y,n)*omega(x,y);
	
	//structure for 2nd boundary problem
//     return 	phi(x,y,n)-omega(x,y)*
//             ((omega(x+diff_step,y)-omega(x-diff_step,y))*(phi(x+diff_step,y,n)-phi(x-diff_step,y,n))
//              +(omega(x,y+diff_step)-omega(x,y-diff_step))*(phi(x,y+diff_step,n)-phi(x,y-diff_step,n)))*
//             glob_delta*glob_delta*0.25;

}

__device__ double right_part_f(double x, double y)
{
    return 12.*(y*y*(x*x*x*x-1.) + x*x*(y*y*y*y-1.));
}



__host__ double Hphi (double x, double y, int n)
{
	//polynomial basis
//     return pow(x,n%N)*pow(y,n/N);

	double cubic_stepx = (X1-X0)/(double)(N-1);
	double cubic_stepy = (Y1-Y0)/(double)(N-1);
	
	return host_f_B_3((N-1)/(X1-X0)*(x-X0-cubic_stepx*(double)(n%(N))))*
           host_f_B_3((N-1)/(Y1-Y0)*(y-Y0-cubic_stepy*(double)(n/(N))));
	
}
__host__ double Homega(double x, double y) 
{
    return (x-X0)*(x-X1)*(y-Y0)*(y-Y1);
}
__host__ double Hstructure(double x, double y, int n)
{
	//structure for 1st boundary problem
    return Hphi(x,y,n)*Homega(x,y);
	
	//structure for 2nd boundary problem
// 	return 	Hphi(x,y,n)-Homega(x,y)*
//             ((Homega(x+diff_step,y)-Homega(x-diff_step,y))*(Hphi(x+diff_step,y,n)-Hphi(x-diff_step,y,n))
//              +(Homega(x,y+diff_step)-Homega(x,y-diff_step))*(Hphi(x,y+diff_step,n)-Hphi(x,y-diff_step,n)))*
//             glob_delta*glob_delta*0.25;
}